#include "matrix.cuh"

Matrix *new_unified_matrix(unsigned height, unsigned width) {
    ElementType *device_array;
    hipMalloc(&device_array, height * width * sizeof(ElementType));
    Matrix *unified_matrix;
    hipMallocManaged(&unified_matrix, sizeof(Matrix));
    unified_matrix->dim = 2;
    unified_matrix->height = height;
    unified_matrix->width = width;
    unified_matrix->data = device_array;
    return unified_matrix;
}

Matrix *new_unified_matrix(unsigned batch, unsigned height, unsigned width) {
    ElementType *device_array;
    hipError_t err = hipMalloc(&device_array, batch * height * width * sizeof(ElementType));
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << "\n";
        exit(-1);
    }
    Matrix *unified_matrix;
    hipMallocManaged(&unified_matrix, sizeof(Matrix));
    unified_matrix->dim = 3;
    unified_matrix->batch = batch;
    unified_matrix->height = height;
    unified_matrix->width = width;
    unified_matrix->data = device_array;
    return unified_matrix;
}

Matrix *host_device_by_matrix(std::vector<ElementType> &host, unsigned height, unsigned width) {
    assert(host.size() == height * width);
    Matrix *device_matrix = new_unified_matrix(height, width);
    hipMemcpy(device_matrix->data, host.data(), host.size() * sizeof(ElementType), hipMemcpyHostToDevice);
    return device_matrix;
}

Matrix *host_device_by_matrix(std::vector<ElementType> &host, unsigned batch, unsigned height, unsigned width) {
    assert(host.size() == height * width * batch);
    Matrix *device_matrix = new_unified_matrix(batch, height, width);
    hipMemcpy(device_matrix->data, host.data(), host.size() * sizeof(ElementType), hipMemcpyHostToDevice);
    return device_matrix;
}

void free_matrix(Matrix *matrix) {
    matrix->~Matrix();
    hipFree(matrix);
}